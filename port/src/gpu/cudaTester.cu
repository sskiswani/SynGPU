#include "hip/hip_runtime.h"
#include "helpers.h"
#include "utility.h"

#pragma clang diagnostic push
#pragma ide diagnostic ignored "CannotResolve"
__global__ void print_kernel( TArray2<int> *dtester ) {
    if (blockIdx.x == threadIdx.x)
        ((*dtester)(threadIdx.x, blockIdx.x)) = 1337;
}

#pragma clang diagnostic pop

int main( int argc, char *argv[] ) {
    //~ Prepare device.
    hipDeviceProp_t prop;
    int whichDevice;
    HANDLE_ERROR(hipGetDevice(&whichDevice));
    HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));
    struct Timer timer;


    TArray2<int> tester(10, 10);
    for (int y = 0, i = 0; y < 10; ++y) {
        for (int x = 0; x < 10; ++x, ++i) {
            tester(x, y) = i;
        }
    }

    std::cerr << "Tester:\n" << tester;

    std::cerr << "Copying to device..." << std::endl;
    TArray2<int> *dtester = tester.CopyToDevice();
    hipDeviceSynchronize();

    std::cerr << "Running kernel now..." << std::endl;

    print_kernel <<<10, 10>>> ((TArray2<int> *) (dtester));
    hipDeviceSynchronize();

    std::cerr << "Copying to host..." << std::endl;

    TArray2<int> helper;
    HANDLE_ERROR(hipMemcpy(&helpet, dtester, sizeof(TArray2<int>), hipMemcpyDeviceToHost));

    int* t_data = tester._data;
    HANDLE_ERROR(hipMemcpy(tester._data, helper.data, tester.Bytes(), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();


    std::cerr << "Tester:\n" << tester;

}
