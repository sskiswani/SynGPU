//#include <stdlib.h>
//#include <math.h>
//#include <time.h>
//#include <string>
//#include <iostream>
//#include <fstream>
//#include <sstream>

#include <iostream>
#include "utility.h"
#include "cu_synfire.cuh"
#include "cuda_utils.h"


int main(int argc, char *argv[]) {
    std::cout << "Welcome to SynfireGrowth powered by NVIDIA - CUDA.\n" << std::endl;

    //~ Prepare device.
    hipDeviceProp_t prop;
    int whichDevice;
    HANDLE_ERROR( hipGetDevice( &whichDevice) );
    HANDLE_ERROR( hipGetDeviceProperties( &prop, whichDevice) );
    struct Timer timer;

    // TODO: Command line arguments

    timer.Start();
    CUSynfire synfire = CUSynfire::CreateCUSynfire();
    timer.Stop();
    std::cout << "Initialization time: " << US_TO_MS(timer.Duration()) << " ms." << std::endl;

    synfire.Run();
}
